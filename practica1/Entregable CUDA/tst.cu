#include "hip/hip_runtime.h"
#include "file_system.h"

#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void blur(uint8_t * red, uint8_t * redNewDevice, int width, int height){
    printf("Here\n");
    for(int i = 0; i < width * height; i++) redNewDevice[i] = red[i];
}
int main(void) {

    char * imgname = "LittleRGB.jpg"; //argv[1]; 		// "GrandImg.jpg";//"img2.jpg";//"icon.png";//"GrandImg.jpg";

     // Image Lecture
    Image * img =  (Image *)malloc(sizeof(Image));
    if(!readImage(imgname, img)){
        printf("%s: ERROR LEYENDO IMAGEN\n", imgname);
        return 0;
    }

    int arrSize = img->width*img->height*sizeof(uint8_t);
    printf("size:%d\n", arrSize);

    uint8_t * redDevice, * redNewDevice, * redNewHost = (uint8_t * )malloc(arrSize);

    hipMalloc(&redDevice, arrSize);
    hipMalloc(&redNewDevice, arrSize);

    hipMemcpy(redDevice, img->red, arrSize, hipMemcpyHostToDevice);

    blur<<<1, 1>>>(redDevice, redNewDevice, img->width, img->height);	

    hipMemcpy(redNewHost, redNewDevice, arrSize, hipMemcpyDeviceToHost);

    for(int i = 0; i < img->width*img->height; i++) printf("%d  ", img->red[i]);
    printf("\n");
    for(int i = 0; i < img->width*img->height; i++) printf("%d  ", redNewHost[i]);
    printf("\n");

    hipFree(redDevice);
    hipFree(redNewDevice);
    freeImage(img);
    free(redNewHost);
    
	return 0;
}
