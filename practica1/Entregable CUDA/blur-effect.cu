#include "hip/hip_runtime.h"
// Compile with nvcc blur-effect.cu -o blur-effect -I /usr/local/cuda/samples/common/inc -lm to include hip/hip_runtime_api.h

#include "file_system.h"
#define M_PI 3.14159265358979323846
#include <sys/time.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/**
 * bsize represents the block size of the blockwise sistem
 * threads represent the number of threads of the algorithm
 * ksize represents the kernel size
 * sigma represents the standard deviation of the kernel
 */ 

int bsize = 32,  ksize = 15;
double threadsPerCore = 2;
double sigma = 10;
Image *img, *newImg;
double * * kernel;

double * * gaussianKernel(int size){
    double mean = size/2;
    double * * kernel = (double * *)malloc(size * sizeof(double *));
    for(int i = 0; i < size; i++){
        kernel[i] = (double *)malloc(size * sizeof(double));
        for(int j = 0; j < size; j++){
            kernel[i][j] = exp( -0.5 * (pow((i-mean)/sigma, 2.0) + pow((j-mean)/sigma,2.0)))
                         / (2 * M_PI * sigma * sigma);
        }
    }
    return kernel;
}


__global__  
void blur(uint8_t * color, uint8_t * newColor, double * kernel, int ksize, int imgWidth, int imgHeight, double iterations){
    
    int i;

    // Put on shared memory the kernel
    extern __shared__ double skernel[];

    for (i = threadIdx.x; i < ksize*ksize; i += blockDim.x) {
        skernel[i] = kernel[i];
    }
    __syncthreads();

    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int x, y, k, l;
    double sum, cColor;
    for (i = iterations*index; i < iterations*(index + 1) && i < imgWidth*imgHeight; i++){
        x = i % imgWidth;
        y = i / imgWidth;
        sum = cColor = 0;
        // For each location in kernel
        for(k = ksize/-2; k <= ksize/2; k++) for(l = ksize/-2; l <= ksize/2; l++) {
            if(y + k < 0 || y + k >= imgHeight || x + l < 0 || x + l >= imgWidth) continue;
            
            cColor += color[(y + k)*imgWidth + x + l] * skernel[(k + ksize/2)*ksize + l + ksize/2];
            // For normalization
            sum += skernel[(k + ksize/2)*ksize + l + ksize/2];
        }        

        newColor[i] = cColor/sum;
    }
}

int main(int argc, char **argv) {

    
    // For time stamp
    struct timeval start, stop, diff;
    gettimeofday(&start, NULL);
    
    char * imgname = argv[1]; 		// "GrandImg.jpg";//"img2.jpg";//"icon.png";//"GrandImg.jpg";
    char * newImgName = argv[2];	//"GrandImgBlur.jpg";//"img2Blur.jpg";//"iconBlur.png";//"GrandImgBlur.jpg";
    ksize = atoi(argv[3]);
    threadsPerCore = atof(argv[4]);
    //printf("Ejecutando programa con:\n\t- %d threads\n\t- %d tamaño del kernel\n\t- %d tamaño de bloque\n\t- %.3f sigma\n", threads, ksize, bsize, sigma);
 
    // Even kernel size
    if((ksize & 1) == 0){
        printf("Detectado tamaño de kernel par (%d). Solo se pueden usar tamaños impares de kernel por lo que se usará %d\n", ksize, ksize + 1);
        ksize++;
    }

    // Image Lecture
    Image * img =  (Image *)malloc(sizeof(Image));
    if(!readImage(imgname, img)){
        printf("%s: ERROR LEYENDO IMAGEN\n", imgname);
        return 0;
    }

    //printf("Imagen %s cargada\n", imgname);

    // Kernel
    kernel = gaussianKernel(ksize);
    
    ///////////////////////////////////////Parallel setup/////////////////////////////////////

    //CARD INFO
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int BLOCKS = 2 * deviceProp.multiProcessorCount, THREADS = threadsPerCore * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);

    // Alloc Kernel in memory
    double * kDevice;
    hipMalloc(&kDevice, ksize*ksize*sizeof(double));
    for(int i = 0; i < ksize; i++){
        hipMemcpy(kDevice + i*ksize, kernel[i], ksize*sizeof(double), hipMemcpyHostToDevice);
    }

    // Memory 
    int arrSize = img->width*img->height*sizeof(uint8_t);
    uint8_t * colorDevice, * colorNewDevice;
    
    hipMalloc(&colorDevice, arrSize);
    hipMalloc(&colorNewDevice, arrSize);

    double iterations = (1.0*img->width*img->height)/(BLOCKS * THREADS);
    
    if(BLOCKS * THREADS > img->width*img->height) iterations = 1;
    int sharedMemory = ksize*ksize*sizeof(double);

    printf("Size: %d, iterations: %f\n", img->width*img->height, iterations * BLOCKS * THREADS);

    printf("Running with %d threads, %d BLocks, %d ksize, %f iterations\n", THREADS, BLOCKS, ksize, iterations);

    uint8_t * currentPointer;

    for (int i = 0; i < 3; i++){
        printf("Running color %d\n", i);
        // Alloc Image in memory;
        switch(i){
            case 0: currentPointer = img->red; break;
            case 1: currentPointer = img->blue; break;
            case 2: currentPointer = img->green; break;
        }

        hipMemcpy(colorDevice, currentPointer, arrSize, hipMemcpyHostToDevice);

        // Execution
        blur<<<BLOCKS, THREADS, sharedMemory>>>(colorDevice, colorNewDevice, kDevice, ksize, img->width, img->height, iterations);	

        // Copy results
        hipMemcpy(currentPointer, colorNewDevice, arrSize, hipMemcpyDeviceToHost);
        
    }

    // Save new image
    //printf("Guardando imagen\n");
    writeImage(img, newImgName);

    // Free
    hipFree(colorDevice);
    hipFree(colorNewDevice);
    freeImage(img);

    //Stadistics
    gettimeofday(&stop, NULL);
    timersub(&stop, &start, &diff);
    printf("Tiempo: %ld.%06ld\n", (long int) diff.tv_sec, (long int) diff.tv_usec);

    return 0;
}
