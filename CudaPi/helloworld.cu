
#include <hip/hip_runtime.h>
#include <stdio.h> 
 
__global__ void calcpi(long long iteraciones, int nhilos, double *pi){
	
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	double tmppi = 0;
	//printf("Hilo: %d \n", index);
	long long i = (iteraciones/nhilos) * index, end = (iteraciones/nhilos) * (index + 1);
	
	do{
	
		if((i & 1) == 0) tmppi += (double)(4.0/((i << 1) | 1)); // (long double)(4.0/((2 * i) + 1));
		else tmppi -=  (double)(4.0/((i << 1) | 1)); // (long double)(4.0/((2 * i) + 1));
		i++;
		
	}while(i < end);
	pi[index] = tmppi;
}
int main(void) {
	long long iterations=2e10;
	int nhilos=1280;
	
	double *pi, *pi_device;
	
	hipMalloc(&pi_device, 1280*sizeof(double));
	
	pi=(double *)malloc(1280*sizeof(double));


	//cudaMemcpy(ite_device,iterations, sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(nhilos_device,nhilos, sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(pi_device,pi, 1280*sizeof(double), cudaMemcpyHostToDevice);

	calcpi<<<5,256>>>(iterations,nhilos,pi_device);	

	hipMemcpy(pi,pi_device, 1280*sizeof(double), hipMemcpyDeviceToHost);
	
	double out;
	for(int i=0; i<1280; i++){	
		out+=pi[i];
	}
	printf("PI: %.20f. \n", out);
	
	free(pi);
    hipFree(pi_device);
    
	return 0;
}
