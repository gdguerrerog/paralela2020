#include "hip/hip_runtime.h"
// Compile with nvcc calcPiCUDA.cu -o calcPi -I /usr/local/cuda/samples/common/inc to include hip/hip_runtime_api.h

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define ITERATIONS 8e9

__global__ void
caclPi(double * vals, long long size)
{
    int indx = blockDim.x * blockIdx.x + threadIdx.x;

    double pi = 0;
    for(long long i = indx * size; i < (indx + 1)*size; i++){
        if((i & 1) != 0) pi -= 4.0/((i << 1) | 1); // (double)(4.0/((2 * i) + 1));
        else pi += 4.0/((i << 1) | 1);
    }

    vals[indx] = pi;
}

/**
 * Host main routine
 */
int
main(void)
{

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int BLOCKS = deviceProp.multiProcessorCount, THREADS = 2 * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);

    int size = BLOCKS * THREADS;

    double * values = (double *)malloc(size * sizeof(double));

    double * d_values = NULL;

    hipMalloc(&d_values, size * sizeof(double));

    long long iterations_per_thread = ITERATIONS/size;

    printf("Running with %d multiprocessor, and %d threads for MP.\nIterations: %.0f, iterations per thread: %lli\n", 
        BLOCKS, THREADS, ITERATIONS, iterations_per_thread);

    caclPi<<<BLOCKS, THREADS>>>(d_values, iterations_per_thread);

    hipMemcpy(values, d_values, size * sizeof(double), hipMemcpyDeviceToHost);

    double pi = 0;

    for(int i = 0; i < size; i++){
        pi += values[i];
    }

    printf("pi: %.20f\n", pi);


    hipFree(d_values);
    free(values);

    return 0;
}

